// vector_add.cu

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__global__ void vecAdd(const float *a, const float *b, float *c, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    c[i] = a[i] + b[i];
}

int main()
{
  const int N = 1 << 20; // 1 M 要素
  const size_t bytes = N * sizeof(float);

  // ホスト側メモリ確保
  float *h_a = (float *)malloc(bytes);
  float *h_b = (float *)malloc(bytes);
  float *h_c = (float *)malloc(bytes);

  // データ初期化
  for (int i = 0; i < N; ++i)
  {
    h_a[i] = 1.0f;
    h_b[i] = 2.0f;
  }

  // デバイス側メモリ確保
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // ホスト → デバイス
  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  // カーネル起動
  int threads = 256;
  int blocks = (N + threads - 1) / threads;
  vecAdd<<<blocks, threads>>>(d_a, d_b, d_c, N);

  // デバイス → ホスト
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  // 結果検証
  bool ok = true;
  for (int i = 0; i < N; ++i)
  {
    if (h_c[i] != 3.0f)
    {
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "All good 🎉" : "Mismatch! ❌");

  // 後片付け
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(h_a);
  free(h_b);
  free(h_c);
  return ok ? 0 : 1;
}
